#include "hip/hip_runtime.h"
/* common utility functions
  1. GPU info
  2. GPU memory status
  3. next2357
*/

#include <cstdlib>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <stdio.h>
//#include <thrust/extrema.h>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>

#include "datatype.h"
void GPU_info()
{

  printf("Starting... \n");
  int deviceCount = 0;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);
  if (error_id != hipSuccess)
  {
    printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
    exit(EXIT_FAILURE);
  }

  if (deviceCount == 0)
    printf("There is no available device that support CUDA\n");
  else
    printf("Detected %d CUDA capable device(s)\n", deviceCount);

  int dev, driverVersion = 0, runtimeVersion = 0;

  dev = 0;

  printf("Input the device index:");
  std::cin >> dev;
  hipSetDevice(dev);

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, dev);
  printf("Device %d: %s\n", dev, deviceProp.name);

  hipDriverGetVersion(&driverVersion);
  hipRuntimeGetVersion(&runtimeVersion);
  printf("CUDA Driver Version / Runtime Version       %d.%d / %d.%d\n",
         driverVersion / 1000, driverVersion % 1000, runtimeVersion / 1000, runtimeVersion % 1000);
  printf("CUDA Capability Major/Minor version number: %d.%d\n", deviceProp.major, deviceProp.minor);
  printf("Total amount of global memory:              %.2f MBytes\n", (float)deviceProp.totalGlobalMem / (pow(1024.0, 2)));
  printf("GPU clock rate:                             %.0f MHz\n", deviceProp.clockRate * 1e-3f);
  printf("Memory clock rate:                          %.0f MHz\n", deviceProp.memoryClockRate * 1e-3f);
  printf("Memory Bus Width:                           %d-bit\n", deviceProp.memoryBusWidth);

  if (deviceProp.l2CacheSize)
  {
    printf("L2 Cache Size:                          %d bytes\n", deviceProp.l2CacheSize);
  }
  printf("Total amount of constant memory:            %lu bytes\n", deviceProp.totalConstMem);
  printf("Total amount of shared memory per block:    %lu bytes\n", deviceProp.sharedMemPerBlock);
  printf("Total number of registers available per block: %d\n", deviceProp.regsPerBlock);
  printf("Warp size:                                  %d\n", deviceProp.warpSize);
  printf("Number of multiprocessors:                  %d\n", deviceProp.multiProcessorCount);
  printf("Maximum number of threads per multiprocessor: %d\n", deviceProp.maxThreadsPerMultiProcessor);
  //printf("Maximum number of blocks per multiprocessor: %d\n",deviceProp.maxBlocksPerMultiProcessor);
  printf("Maximum number of thread per block:          %d\n", deviceProp.maxThreadsPerBlock);
  printf("Maximum sizes of each dimension of a block: %d x %d x %d\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
  printf("Maximum sizes of each dimension of a grid:  %d x %d x %d\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1],
         deviceProp.maxGridSize[2]);
  printf("Maximum memory pitch:                       %lu bytes\n", deviceProp.memPitch);
}

int next235beven(int n, int b)
// finds even integer not less than n, with prime factors no larger than 5
// (ie, "smooth") and is a multiple of b (b is a number that the only prime
// factors are 2,3,5). Adapted from fortran in hellskitchen. Barnett 2/9/17
// changed INT64 type 3/28/17. Runtime is around n*1e-11 sec for big n.
// added condition about b Melody 05/31/20
{
  if (n <= 2)
    return 2;
  if (n % 2 == 1)
    n += 1;          // even
  int nplus = n - 2; // to cancel out the +=2 at start of loop
  int numdiv = 2;    // a dummy that is >1
  while ((numdiv > 1) || (nplus % b != 0))
  {
    nplus += 2; // stays even
    numdiv = nplus;
    while (numdiv % 2 == 0)
      numdiv /= 2; // remove all factors of 2,3,5...
    while (numdiv % 3 == 0)
      numdiv /= 3;
    while (numdiv % 5 == 0)
      numdiv /= 5;
  }
  return nplus;
}

void show_mem_usage()
{
  // show memory usage of GPU

  size_t free_byte;

  size_t total_byte;

  checkCudaErrors(hipMemGetInfo(&free_byte, &total_byte));

  double free_db = (double)free_byte;

  double total_db = (double)total_byte;

  double used_db = total_db - free_db;

  printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n",
         used_db / 1024.0 / 1024.0, free_db / 1024.0 / 1024.0, total_db / 1024.0 / 1024.0);
}